#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <fstream>
#include <ctime>
#include <chrono>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define MAX_THREAD 1024

using namespace std;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

// this method is using n as block width for better global memory coalescing
__global__ void compute_LU(int i, int B, int B_num, int n, double *A)
{
  __shared__ double multiplier[1];

  int x = (B * (blockIdx.x % B_num)) + threadIdx.x;
  int y = blockIdx.x / B_num + i;

  // calculate multiplier
  if (x == 0)
  {
    multiplier[0] = A[y * n + (i - 1)] / A[(i - 1) * n + (i - 1)];
    __syncthreads();
  }
  
  // update row
  A[y * n + (x + (i - 1))] = A[y * n + (x + (i - 1))] - (multiplier[0] * A[(i - 1) * n + (x + (i - 1))]); 

  // write back multiplier
  A[y * n + (i - 1)] = multiplier[0];
  
}

void print_matrix(double *A, int N, int n)
{
  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < N; ++j)
    {
      printf("%.5f ", A[i * n + j]);
    }
    printf("\n");
  }
}

int main(int argc, char **argv)
{
  if (argc != 3)
  {
    fprintf(stderr, "must provide exactly 2 arguments N output_filename\n");
    return 1;
  }
  typedef std::chrono::milliseconds ms;
  auto total_starttime = duration_cast<ms>(system_clock::now().time_since_epoch()).count();

  // parsing argument
  int N = atoi(argv[1]);
  char *out_filename = argv[2];

  // generate matrix
  // srand((unsigned)time(NULL));
  int n = N, B = N, B_num = 1;
  // if row size >= max threads in a block then do padding
  if (N >= MAX_THREAD)
  {
    B = MAX_THREAD;
    n = (B * (N / B)) + ((N % B == 0)? 0 : B);
    printf("N %d\n",N);
    printf("n %d\n",n);
    B_num = n / B;
  }
  printf("init success\n");

  double *A = (double *)malloc(n * n * sizeof(double));
  double *L = (double *)malloc(N * N * sizeof(double));

  if ((A == NULL)||(L == NULL))
  {
    printf("malloc failed\n");
    exit(1);
  }


  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < N; ++j)
    {
      A[i * n + j] = 1 + (rand() % 10000);
      L[i * N + j] = 0;
    }
    // ensure diagonally dominant
    A[i * n + i] = A[i * n + i] + 10000 * N;
  }
  printf("alloc mem success\n");

  // do the padding
  if (N >= MAX_THREAD)
  {
    for (int i = N; i < n; ++i)
    {
      for (int j = 0; j < n; ++j)
      {
        A[i * n + j] = 1000 + i + j;
      }
      A[i * n + i] = A[i * n + i] + 10000 * n;
    }
    for (int j = N; j < n; ++j)
    {
      for (int i = 0; i < n - N; ++i)
      {
        A[i * n + j] = 1000 + i + j;
      }
    }
  }
  printf("padding success\n");

  // print matrix before lu factorization
  if (N < 11)
  {
    printf("the matrix before lu factorization is\n");
    print_matrix(A, N, n);
  }

  // allocate and copy memory to device
  double *device_A;
  hipMalloc(&device_A, n * n * sizeof(double));
  hipMemcpy(device_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);

  // LU factorization
  for (int i = 1; i < N; ++i)
  {
    compute_LU<<<(N - i) * B_num, B - (i - 1)>>>(i, B, B_num, n, device_A); 
  }

  // copy result back to host
  hipMemcpy(A, device_A, n * n * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(device_A);

  // extract L and U
  for (int i = 1; i < N; ++i)
  {
    for (int j = i - 1; j >= 0; --j)
    {
      L[i * N + j] = A[i * n + j];
      A[i * n + j] = 0;
    }
  }

  // assign 1 to diagonal of L
  for (int i = 0; i < N; ++i)
  {
    L[i * N + i] = 1;
  }

  // print outcome
  if (N < 11)
  {
    printf("the lu factorization outcome is\n");
    printf("U is\n");
    print_matrix(A, N, n);
    printf("L is\n");
    print_matrix(L, N, N);
  }

  // write result to output file
  ofstream out_file(out_filename);
  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < N; ++j)
    {
      out_file.write((char *)&A[i * n + j], sizeof(double));
    }
  }
  for (int i = 0; i < N * N; ++i)
  {
    out_file.write((char *)&L[i], sizeof(double));
  }
  out_file.close();
  free(A);
  free(L);

  // calculate total spent time
  auto total_endtime = duration_cast<ms>(system_clock::now().time_since_epoch()).count();
  printf("total time spent for blocked lu %ld ms\n", (total_endtime - total_starttime));
}